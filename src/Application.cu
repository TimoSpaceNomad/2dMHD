﻿#include "Application.cuh"

#include <ctime>

#include "Params.h"
#include "Solver.cuh"

void cuda_main(const std::filesystem::path& outputDir) {
    hipSetDevice(0);

    std::cout << "Printing parameters..." << std::endl;
    mhd::parameters::ParametersPrint();
    mhd::parameters::ParametersSave(outputDir);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    {
        mhd::Solver solver;

        // Initial Conditions
        solver.fillNormally(std::time(nullptr));

        // Initial Energy and Time Step
        solver.updateEnergies();
        solver.updateTimeStep();

        // Zero Data Output
        solver.saveData(outputDir);
        solver.printCurrentParams();

        // Main Cycle of the Program
        while (solver.shouldContinue()) {
            // Saving fields from previous timelayer
            solver.saveOldFields();

            // Time Integration Scheme
            // Two-step Scheme

            // First step
            solver.calcKineticRigthPart();
            solver.timeSchemeKin();

            solver.calcMagneticRightPart();
            solver.timeSchemeMag();

            solver.updateStream();
            solver.updateCurrent();

            // Second step
            solver.calcKineticRigthPart();
            solver.timeSchemeKin();

            solver.calcMagneticRightPart();
            solver.timeSchemeMag();

            solver.updateStream();
            solver.updateCurrent();

            // Update Parameters (Energy and Time Step)
            solver.updateEnergies();
            solver.updateTimeStep();

            // Data Output
            solver.timeStep();
            solver.saveData(outputDir);
            solver.printCurrentParams();
        }
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    std::cout << std::endl;

    // CalculationTime Output
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Calculation time: " << elapsedTime / 1000 << std::endl;

    // Memory Clearing
    hipEventDestroy(stop);
    hipEventDestroy(start);
}
