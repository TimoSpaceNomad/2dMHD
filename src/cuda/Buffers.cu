#include "cuda/Buffers.cuh"

#include "cuda/KernelCaller.cuh"

namespace mhd {

// CpuDoubleBuffer2D functions definitions
CpuDoubleBuffer2D::CpuDoubleBuffer2D()
    : _buffer(nullptr), _sideLength(0), _bufferSize(0) {}

CpuDoubleBuffer2D::CpuDoubleBuffer2D(unsigned int sideLength)
    : _sideLength(sideLength) {
    _bufferSize = _sideLength * _sideLength * sizeof(double);

    CUDA_CALL(
        hipHostAlloc((void**)&_buffer, _bufferSize, hipHostMallocDefault));
}

CpuDoubleBuffer2D::~CpuDoubleBuffer2D() {
    CUDA_CALL(hipHostFree(_buffer));
}

void CpuDoubleBuffer2D::clear() {
    memset(_buffer, 0x0, _bufferSize);
}

void CpuDoubleBuffer2D::copyToDevice(hipStream_t& stream, double* dst) const {
    CUDA_CALL(hipMemcpyAsync(dst, _buffer, _bufferSize, hipMemcpyHostToDevice,
                              stream));
}

void CpuDoubleBuffer2D::copyFromDevice(hipStream_t& stream,
                                       const double* src) {
    if (src != nullptr) {
        CUDA_CALL(hipMemcpyAsync(_buffer, src, _bufferSize,
                                  hipMemcpyDeviceToHost, stream));
    } else {
        std::cerr << "Copying Error from device: Source buffer is nullptr!"
                  << std::endl;
    }
}

// GpuDoubleBuffer2D functions definitions
GpuDoubleBuffer2D::GpuDoubleBuffer2D()
    : _buffer(nullptr), _sideLength(0), _bufferSize(0) {}

GpuDoubleBuffer2D::GpuDoubleBuffer2D(unsigned int sideLength)
    : _sideLength(sideLength) {
    _bufferSize = _sideLength * _sideLength * sizeof(double);

    CUDA_CALL(hipMalloc((void**)&_buffer, _bufferSize));
}

GpuDoubleBuffer2D::~GpuDoubleBuffer2D() {
    CUDA_CALL(hipFree(_buffer));
}

void GpuDoubleBuffer2D::clear(hipStream_t& stream) {
    CUDA_CALL(hipMemsetAsync(_buffer, 0x0, _bufferSize, stream));
}

void GpuDoubleBuffer2D::copyToHost(hipStream_t& stream, double* dst) const {
    CUDA_CALL(hipMemcpyAsync(dst, _buffer, _bufferSize, hipMemcpyDeviceToHost,
                              stream));
}

void GpuDoubleBuffer2D::copyFromHost(hipStream_t& stream, const double* src) {
    CUDA_CALL(hipMemcpyAsync(_buffer, src, _bufferSize, hipMemcpyHostToDevice,
                              stream));
}

void GpuDoubleBuffer2D::copyToDevice(hipStream_t& stream, double* dst) const {
    CUDA_CALL(hipMemcpyAsync(dst, _buffer, _bufferSize,
                              hipMemcpyDeviceToDevice, stream));
}

void GpuDoubleBuffer2D::copyFromDevice(hipStream_t& stream,
                                       const double* src) {
    CUDA_CALL(hipMemcpyAsync(_buffer, src, _bufferSize,
                              hipMemcpyDeviceToDevice, stream));
}

// GpuComplexBuffer2D functions definitions
GpuComplexBuffer2D::GpuComplexBuffer2D()
    : _buffer(nullptr), _sideLength(0), _bufferSize(0) {}

GpuComplexBuffer2D::GpuComplexBuffer2D(unsigned int sideLength)
    : _sideLength(sideLength) {
    _bufferSize =
        (_sideLength / 2 + 1) * _sideLength * sizeof(hipfftDoubleComplex);

    CUDA_CALL(hipMalloc((void**)&_buffer, _bufferSize));
}

GpuComplexBuffer2D::~GpuComplexBuffer2D() {
    CUDA_CALL(hipFree(_buffer));
}

void GpuComplexBuffer2D::clear(hipStream_t& stream) {
    CUDA_CALL(hipMemsetAsync(_buffer, 0x0, _bufferSize, stream));
}

void GpuComplexBuffer2D::copyToHost(hipStream_t& stream,
                                    hipfftDoubleComplex* dst) const {
    CUDA_CALL(hipMemcpyAsync(dst, _buffer, _bufferSize, hipMemcpyDeviceToHost,
                              stream));
}

void GpuComplexBuffer2D::copyFromHost(hipStream_t& stream,
                                      const hipfftDoubleComplex* src) {
    CUDA_CALL(hipMemcpyAsync(_buffer, src, _bufferSize, hipMemcpyHostToDevice,
                              stream));
}

void GpuComplexBuffer2D::copyToDevice(hipStream_t& stream,
                                      hipfftDoubleComplex* dst) const {
    CUDA_CALL(hipMemcpyAsync(dst, _buffer, _bufferSize,
                              hipMemcpyDeviceToDevice, stream));
}

void GpuComplexBuffer2D::copyFromDevice(hipStream_t& stream,
                                        const hipfftDoubleComplex* src) {
    CUDA_CALL(hipMemcpyAsync(_buffer, src, _bufferSize,
                              hipMemcpyDeviceToDevice, stream));
}

// GpuStateBuffer2D functions definitions

GpuStateBuffer2D::GpuStateBuffer2D()
    : _buffer(nullptr), _sideLength(0), _bufferSize(0) {}

GpuStateBuffer2D::GpuStateBuffer2D(unsigned int sideLength)
    : _sideLength(sideLength) {
    _bufferSize = (_sideLength / 2 + 1) * _sideLength * sizeof(hiprandState);

    CUDA_CALL(hipMalloc((void**)&_buffer, _bufferSize));
}

GpuStateBuffer2D::~GpuStateBuffer2D() {
    CUDA_CALL(hipFree(_buffer));
}
}  // namespace mhd

namespace graphics {

// CpuFloatBuffer functions definitions

CpuFloatBuffer::CpuFloatBuffer()
    : _buffer(nullptr), _bufferLength(0), _bufferSize(0) {}

CpuFloatBuffer::CpuFloatBuffer(unsigned int bufferLength)
    : _bufferLength(bufferLength) {
    _bufferSize = _bufferLength * sizeof(float);

    CUDA_CALL(
        hipHostAlloc((void**)&_buffer, _bufferSize, hipHostMallocDefault));
}

CpuFloatBuffer::~CpuFloatBuffer() {
    CUDA_CALL(hipHostFree(_buffer));
}

void CpuFloatBuffer::clear() {
    memset(_buffer, 0x0, _bufferSize);
}

void CpuFloatBuffer::copyToDevice(hipStream_t& stream, float* dst) const {
    CUDA_CALL(hipMemcpyAsync(dst, _buffer, _bufferSize, hipMemcpyHostToDevice,
                              stream));
}

void CpuFloatBuffer::copyFromDevice(hipStream_t& stream, const float* src) {
    CUDA_CALL(hipMemcpyAsync(_buffer, src, _bufferSize, hipMemcpyDeviceToHost,
                              stream));
}

// GpuFloatBuffer functions definitions

GpuFloatBuffer::GpuFloatBuffer()
    : _buffer(nullptr), _bufferLength(0), _bufferSize(0) {}

GpuFloatBuffer::GpuFloatBuffer(unsigned int bufferLength)
    : _bufferLength(bufferLength) {
    _bufferSize = _bufferLength * sizeof(float);

    CUDA_CALL(hipMalloc((void**)&_buffer, _bufferSize));
}

GpuFloatBuffer::~GpuFloatBuffer() {
    hipFree(_buffer);
}

void GpuFloatBuffer::clear(hipStream_t& stream) {
    CUDA_CALL(hipMemsetAsync(_buffer, 0x0, _bufferSize, stream));
}

void GpuFloatBuffer::copyToDevice(hipStream_t& stream, float* dst) const {
    CUDA_CALL(hipMemcpyAsync(dst, _buffer, _bufferSize, hipMemcpyHostToDevice,
                              stream));
}

void GpuFloatBuffer::copyFromDevice(hipStream_t& stream, const float* src) {
    CUDA_CALL(hipMemcpyAsync(_buffer, src, _bufferSize, hipMemcpyDeviceToHost,
                              stream));
}

// CpuPixelBuffer2D functions definitions

CpuPixelBuffer2D::CpuPixelBuffer2D()
    : _buffer(nullptr), _sideLength(0), _bufferSize(0), _channels(0) {}

CpuPixelBuffer2D::CpuPixelBuffer2D(unsigned int sideLength,
                                   unsigned int channels)
    : _sideLength(sideLength), _channels(channels) {
    _bufferSize = _sideLength * _sideLength * _channels * sizeof(unsigned char);

    CUDA_CALL(
        hipHostAlloc((void**)&_buffer, _bufferSize, hipHostMallocDefault));
}

CpuPixelBuffer2D::~CpuPixelBuffer2D() {
    CUDA_CALL(hipHostFree(_buffer));
}

void CpuPixelBuffer2D::clear() {
    memset(_buffer, 0x0, _bufferSize);
}

void CpuPixelBuffer2D::copyToDevice(hipStream_t& stream,
                                    unsigned char* dst) const {
    CUDA_CALL(hipMemcpyAsync(dst, _buffer, _bufferSize, hipMemcpyHostToDevice,
                              stream));
}

void CpuPixelBuffer2D::copyFromDevice(hipStream_t& stream,
                                      const unsigned char* src) {
    CUDA_CALL(hipMemcpyAsync(_buffer, src, _bufferSize, hipMemcpyDeviceToHost,
                              stream));
}

// GPUPixelBuffer2D functions definitions

GpuPixelBuffer2D::GpuPixelBuffer2D()
    : _buffer(nullptr), _sideLength(0), _bufferSize(0), _channels(0) {}

GpuPixelBuffer2D::GpuPixelBuffer2D(unsigned int sideLength,
                                   unsigned int channels)
    : _sideLength(sideLength), _channels(channels) {
    _bufferSize = _sideLength * _sideLength * _channels * sizeof(unsigned char);

    CUDA_CALL(hipMalloc((void**)&_buffer, _bufferSize));
}

GpuPixelBuffer2D::~GpuPixelBuffer2D() {
    hipFree(_buffer);
}

void GpuPixelBuffer2D::clear(hipStream_t& stream) {
    CUDA_CALL(hipMemsetAsync(_buffer, 0x0, _bufferSize, stream));
}

void GpuPixelBuffer2D::copyToHost(hipStream_t& stream,
                                  unsigned char* dst) const {
    CUDA_CALL(hipMemcpyAsync(dst, _buffer, _bufferSize, hipMemcpyDeviceToHost,
                              stream));
}

void GpuPixelBuffer2D::copyFromHost(hipStream_t& stream,
                                    const unsigned char* src) {
    CUDA_CALL(hipMemcpyAsync(_buffer, src, _bufferSize, hipMemcpyHostToDevice,
                              stream));
}
}  // namespace graphics