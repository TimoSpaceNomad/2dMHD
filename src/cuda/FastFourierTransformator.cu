#include "cuda/FastFourierTransformator.cuh"

#include <iostream>

#define CUFFT_CALL(result) \
    checkCufftResult(result, __FUNCTION__, __FILE__, __LINE__)

void checkCufftResult(hipfftResult_t result, const std::string& functionName,
                      const std::string& fileName, int lineNumber) {
    if (result != HIPFFT_SUCCESS) {
        std::cerr << "CUFFT Error " << result << " in " << functionName
                  << " at " << fileName << ":" << lineNumber << std::endl;
    }
}

namespace mhd {
FastFourierTransformator::FastFourierTransformator(unsigned int gridLength) {
    CUFFT_CALL(hipfftPlan2d(&planD2Z, gridLength, gridLength, HIPFFT_D2Z));
    CUFFT_CALL(hipfftPlan2d(&planZ2D, gridLength, gridLength, HIPFFT_Z2D));
}

FastFourierTransformator::~FastFourierTransformator() {
    CUFFT_CALL(hipfftDestroy(planD2Z));
    CUFFT_CALL(hipfftDestroy(planZ2D));
}

void FastFourierTransformator::forwardFFT(double* input,
                                          hipfftDoubleComplex* output) const {
    CUFFT_CALL(hipfftExecD2Z(planD2Z, input, output));
}

void FastFourierTransformator::inverseFFT(hipfftDoubleComplex* input,
                                          double* output) const {
    CUFFT_CALL(hipfftExecZ2D(planZ2D, input, output));
}

void FastFourierTransformator::forward(GpuDoubleBuffer2D& input,
                                       GpuComplexBuffer2D& output) const {
    CUFFT_CALL(hipfftExecD2Z(planD2Z, input.data(), output.data()));
}

void FastFourierTransformator::inverse(GpuComplexBuffer2D& input,
                                       GpuDoubleBuffer2D& output) const {
    CUFFT_CALL(hipfftExecZ2D(planZ2D, input.data(), output.data()));
}

}  // namespace mhd
